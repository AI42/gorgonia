
#include <hip/hip_runtime.h>
#ifdef __cplusplus
extern "C" {
#endif


__global__ void cube64(double* A, int size)
{
	int idx = threadIdx.x;
	if (idx >= size) {
		return;
	}
	A[idx] = A[idx] * A[idx] * A[idx]; 
}
	
#ifdef __cplusplus
}
#endif