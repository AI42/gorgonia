
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>

#ifdef __cplusplus
extern "C" {
#endif


__global__ void square64(double* A, int size)
{
	int i = 0;
	for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size/8; idx += blockDim.x * gridDim.x) {
		printf("A %p | i: %d | idx: %d | size %d | blockIdx.x %d | blockDim.x %d | threadIdx.x %d | gridDim.x %d\n", A, i, idx, size,  blockIdx.x, blockDim.x, threadIdx.x, gridDim.x);
		A[idx] = (double)(idx); 
		i++;
	}
}
	
#ifdef __cplusplus
}
#endif